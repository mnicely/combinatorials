#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

 #include <hip/hip_cooperative_groups.h>
 //  #include <cub/block/block_reduce.cuh>
 #include <hip/hip_runtime.h>
 #include <numeric>
 #include <omp.h>
 #include <thrust/host_vector.h>
 #include <thrust/sort.h>
 #include <thrust/system/cuda/experimental/pinned_allocator.h>
 
 // *************** FOR ERROR CHECKING *******************
 #ifndef CUDA_RT_CALL
 #define CUDA_RT_CALL( call )                                                                                           \
     {                                                                                                                  \
         auto status = static_cast<hipError_t>( call );                                                                \
         if ( status != hipSuccess )                                                                                   \
             fprintf( stderr,                                                                                           \
                      "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "                                        \
                      "with "                                                                                           \
                      "%s (%d).\n",                                                                                     \
                      #call,                                                                                            \
                      __LINE__,                                                                                         \
                      __FILE__,                                                                                         \
                      hipGetErrorString( status ),                                                                     \
                      status );                                                                                         \
     }
 #endif  // CUDA_RT_CALL
 // *************** FOR ERROR CHECKING *******************
 
 /*
  * Factorial required for combinatorial number system
  */
 constexpr size_t factorial( const int &n ) {
     return ( n <= 1 ) ? 1 : ( n * factorial( n - 1 ) );
 }
 
 /*
  * The more we can do at compile time the better
  */
 constexpr unsigned int k_tpb { 512 };
 constexpr unsigned int k_ept { 8 };
 constexpr unsigned int k_numNodes { 10 };
 constexpr unsigned int k_numEdges { 9 };
 constexpr unsigned int k_numVertices { ( k_numNodes * ( k_numNodes - 1 ) ) / 2 };
 constexpr unsigned int k_treeStart { k_numVertices - 1 };
 
 /*
  * Structure hold edge and angle information
  */
 typedef struct fEdgeData_t {
     unsigned int a {};
     unsigned int b {};
 } edgeData;
 
 /*
  * Structure to hold all combos and scores for testing
  */
 typedef struct fgpuData_t {
     size_t       offset {};
     size_t       treesMaxDevice {};
     hipStream_t streams {};
 } gpuData;
 
 /*
  * Constant memory holds read-only data in cached global memory
  */
 __constant__ double c_denominator[k_numEdges];
 __constant__ edgeData c_edges[k_numVertices];
 
 /*
  * Device memory to hold total trees per device
  */
 __device__ unsigned long long int d_totalTreesPerBlock {};
 
 /*
  * Calculate binomial coefficients
  * Care must be taken to ensure arithmetic doesn't
  * exceed what a data type can hold
  */
 __host__ __device__ size_t nchoosek( const unsigned int &numerator, const double &denominator, const int &loops ) {
 
     size_t n { static_cast<size_t>( numerator ) };
     for ( int f = 1; f < loops; f++ )
         n *= ( numerator - f );
     return ( static_cast<size_t>( static_cast<double>( n ) * denominator ) );  // Precalculate
 }
 
 /*
  * This function is find a combination based on a given id
  * It uses the combinatorial number system and produces
  * an answer in lexicographic order
  */
 __host__ __device__ void getTree( const size_t &maxTrees, const size_t &id, unsigned int *combo ) {
 
     size_t n {};
     size_t key { maxTrees - id - 1 };
 #pragma unroll k_numEdges
     for ( int e = 0; e < k_numEdges; e++ ) {
         unsigned int numerator { k_treeStart };
         while ( true ) {
             // The denominator must start at the end of the array
             n = nchoosek( numerator, c_denominator[e], ( k_numEdges - e ) );
             if ( n <= key ) {
                 combo[e] = k_treeStart - numerator;
                 key -= n;
                 break;
             }
             numerator--;
         }
     }
 }
 
 /*
  * This function is used by Kruskal's Minimum Spanning Tree algorithm
  */
 __device__ unsigned int find( const unsigned int &x, unsigned int *parent ) {
 
     if ( parent[x] != x )
         parent[x] = find( parent[x], parent );
     return ( parent[x] );
 }
 
 /*
  * This function is used to generate a sequence on the device
  */
 __device__ void gen_seq( unsigned int *parent ) {
 #pragma unroll  k_numNodes
     for ( int i = 0; i < k_numNodes; i++ ) {
         parent[i] = i;
     }
 }
 
 /*
  * CUDA kernel to determine minimum angle diversity score for a given block.
  * Since we are using CUB BlockRadixSort and grid-stride looping
  * we need to pad the last block in the last grid
  */
 __launch_bounds__( k_tpb ) __global__
     void tdoa( const size_t offset, const size_t treesPerDevice, const size_t maxTrees, const size_t padding ) {
 
     const auto block = cooperative_groups::this_thread_block( );
 
     // Specialize BlockRadixSort for a 1D block of k_tpb threads of type int
     typedef hipcub::BlockReduce<unsigned int, k_tpb> BlockReduce;
 
     // Allocate shared memory for BlockRadixSort
     __shared__ typename BlockReduce::TempStorage temp_storage;
 
     size_t       gid { blockIdx.x * blockDim.x + threadIdx.x };  // Change name
     size_t       stride { blockDim.x * gridDim.x };
     size_t       newTid[k_ept] {};
     unsigned int score[k_ept] {};
 
     for ( size_t tid = gid; tid < padding; tid += stride ) {
 
         // To increase Instruction Level Parallelism (ILP)
         // Each thread will calculate multiple combination scores
         for ( int s = 0; s < k_ept; s++ ) {
             newTid[s] = offset + tid * k_ept + s;
 
             // Ensure only valid combinations are checked
             if ( newTid[s] < treesPerDevice ) {
 
                 // Find tree
                 unsigned int combo[k_numEdges] {};
                 getTree( maxTrees, newTid[s], combo );
 
                 // Determine is chain is valid spanning tree
                 // Use Kruskal's algorithm with each edge weight set to 1
                 // thrust::seq allows us to run Thrust functions
                 // in individual threads
                 unsigned int parent[k_numNodes] {};
                 gen_seq( parent );
 
                 unsigned int cost {};
 #pragma unroll k_numEdges
                 for ( int e = 0; e < k_numEdges; e++ ) {
                     unsigned int findX = find( c_edges[combo[e]].a, parent );
                     unsigned int findY = find( c_edges[combo[e]].b, parent );
                     if ( findX == findY )
                         continue;
                     cost++;
                     parent[findX] = findY;
                 }
 
                 // If the minimum spanning tree has a cost
                 // less than the number of edges, than it
                 // is not a valid spanning tree
                 if ( cost == k_numEdges )
                     score[s] = 1u;
                 else
                     score[s] = 0u;
 
             } else
                 score[s] = 0u;  // For thread ids larger than the number of required combinations
         }
 
         unsigned long long int totalTrees { BlockReduce( temp_storage ).Sum( score ) };
 
         // Once BlockReduce is finished, the sum
         // is now stored in the first address of the score array
         // in the first thread in the block. That value is
         // then stored to global memory to the address pertaining
         // to that blockId
         if ( threadIdx.x == 0 )
             atomicAdd( &d_totalTreesPerBlock, totalTrees );
 
         // We need to sync the block again because we are using grid-stride looping.
         block.sync( );  // Sync block to reuse tempStorage for BlockReduce
     }
 }
 
 int main( int arg, char **argv ) {
 
     // Determine the number of combinations that will be evaluated
     double temp { 1.0 / factorial( k_numEdges ) };
     printf( "%0.15f\n", temp );
     size_t maxTrees { nchoosek( k_numVertices, ( temp ), k_numEdges ) };
     if ( maxTrees >= SIZE_MAX ) {  // maxTrees can't be larger than 4294967295
         std::printf( "combos = %d; chains = %lu\n", k_numVertices, maxTrees );
         throw std::runtime_error( "The number is chains to test is larger than unsigned int can hold.\n" );
     }
     std::printf( "Number of Vertices = %d\n", k_numVertices );
     std::printf( "Max Trees Possible = %lu\n", maxTrees );
 
     // exit( 0 );
 
     // Precompute all possible denominators recipicals
     // Multiplication requires less operations than division
     double denominator[k_numEdges] {};
     for ( int i = k_numEdges; i > 0; i-- )
         denominator[k_numEdges - i] = 1.0 / factorial( i );
 
     // Generate all possible edges of graph in lexicographic order
     edgeData edges[k_numVertices] {};
     int      start { 1 };
     int      idx {};
     for ( int i = 0; i < k_numNodes; i++ ) {
         for ( int j = start; j < k_numNodes; j++ ) {
             edges[idx].a = i;
             edges[idx].b = j;
             idx++;
         }
         start++;
     }
 
     // Get device attributes
     int numDevices {};
     int numSMs {};
     CUDA_RT_CALL( hipGetDeviceCount( &numDevices ) );
     CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );
     std::printf( "Number of GPUs = %d\n", numDevices );
 
     // Padding blocks so we can use CUB BlockReduce in CUDA kernel
     size_t padding { static_cast<size_t>(
         std::ceil( static_cast<double>( maxTrees ) / static_cast<double>( k_ept ) / static_cast<double>( numDevices ) /
                    static_cast<double>( k_tpb ) ) *
         k_tpb ) };
 
     // Will store final results for each GPU using pinned memory
     // Pinned memory is required for async copies
     thrust::host_vector<unsigned long long int, thrust::cuda::experimental::pinned_allocator<unsigned long long int>>
         h_totalTrees( numDevices, 0 );
 
     // Divide up work between all GPUs
     gpuData gpuWork[numDevices] {};
     size_t  chunk { static_cast<size_t>( maxTrees / numDevices ) };
 
     for ( int d = 0; d < numDevices; d++ ) {
         if ( d < ( numDevices - 1 ) ) {
             gpuWork[d].offset         = d * chunk;
             gpuWork[d].treesMaxDevice = ( d + 1 ) * chunk;
         } else {
             gpuWork[d].offset         = d * chunk;
             gpuWork[d].treesMaxDevice = maxTrees;
         }
         printf( "%d: %lu %lu\n", d, gpuWork[d].offset, gpuWork[d].treesMaxDevice );
     }
     printf( "chunk %lu\n", chunk );
 
     // Launch one CPU thread per GPU
     omp_set_num_threads( numDevices );
 #pragma omp parallel
     {
         int ompId { omp_get_thread_num( ) };
 
         // We must set the device in each thread
         // so the correct CUDA context is visible
         CUDA_RT_CALL( hipSetDevice( ompId ) );
         CUDA_RT_CALL( hipStreamCreate( &gpuWork[ompId].streams ) );
 
         // Copy denominators to constant memory
         CUDA_RT_CALL( hipMemcpyToSymbolAsync(HIP_SYMBOL( c_denominator),
                                                denominator,
                                                k_numEdges * sizeof( double ),
                                                0,
                                                hipMemcpyHostToDevice,
                                                gpuWork[ompId].streams ) );
 
         // Copy angles to constant memory
         CUDA_RT_CALL( hipMemcpyToSymbolAsync(HIP_SYMBOL(
             c_edges), edges, k_numVertices * sizeof( edgeData ), 0, hipMemcpyHostToDevice, gpuWork[ompId].streams ) );
     }
 
     // Start timer
     hipEvent_t startEvent { nullptr };
     hipEvent_t stopEvent { nullptr };
     float       elapsed_gpu_ms {};
 
     hipEventCreate( &startEvent, hipEventBlockingSync );
     hipEventCreate( &stopEvent, hipEventBlockingSync );
 
     hipEventRecord( startEvent );
 
 #pragma omp parallel
     {
         int ompId { omp_get_thread_num( ) };
         CUDA_RT_CALL( hipSetDevice( ompId ) );
 
         // The number of blocks launched is based on the number of
         // Streaming Multiprocessor available on the GPU
         dim3 threadPerBlock { k_tpb };
         dim3 blocksPerGrid { static_cast<uint>( 32 * numSMs ) };
 
         void *args[] { &gpuWork[ompId].offset, &gpuWork[ompId].treesMaxDevice, &maxTrees, &padding };
 
         CUDA_RT_CALL( hipLaunchKernel(
             reinterpret_cast<void *>( &tdoa ), blocksPerGrid, threadPerBlock, args, 0, gpuWork[ompId].streams ) );
 
         CUDA_RT_CALL( hipMemcpyFromSymbolAsync( &h_totalTrees[ompId],
                                                  HIP_SYMBOL(d_totalTreesPerBlock),
                                                  sizeof( unsigned long long int ),
                                                  0,
                                                  hipMemcpyDefault,
                                                  gpuWork[ompId].streams ) );
 
         // Sync each stream to ensure data copy is complete
         CUDA_RT_CALL( hipStreamSynchronize( gpuWork[ompId].streams ) );
     }
 
     // Stop timer
     hipEventRecord( stopEvent );
     hipEventSynchronize( stopEvent );
 
     hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent );
     std::printf( "Runtime = %0.2f ms\n", elapsed_gpu_ms );
 
     unsigned long long int h_total { std::accumulate( h_totalTrees.begin( ), h_totalTrees.end( ), 0u ) };
 
     size_t cayley =
         static_cast<size_t>( std::pow( static_cast<double>( k_numNodes ), static_cast<double>( k_numNodes - 2 ) ) );
 
     if ( ( k_numNodes - 1 ) == k_numEdges )
         std::printf( "%llu trees found (%lu)\n", h_total, cayley );
     else
         std::printf( "%llu forests found\n", h_total );
 
         // Data clean up
 #pragma omp parallel
     {
         int ompId { omp_get_thread_num( ) };
         CUDA_RT_CALL( hipSetDevice( ompId ) );
         //  CUDA_RT_CALL( hipFree( gpuWork[ompId].d_totalTreesPerBlock ) );
         CUDA_RT_CALL( hipStreamDestroy( gpuWork[ompId].streams ) );
     }
 
     return ( EXIT_SUCCESS );
 }
 