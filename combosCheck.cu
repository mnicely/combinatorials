#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <cmath>
#include <hip/hip_cooperative_groups.h>
#include <numeric>
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

#include "graph.h"

using uchar = unsigned char;

// *************** FOR ERROR CHECKING *******************
#ifndef CUDA_RT_CALL
#define CUDA_RT_CALL( call )                                                                                           \
    {                                                                                                                  \
        auto status = static_cast<hipError_t>( call );                                                                \
        if ( status != hipSuccess )                                                                                   \
            fprintf( stderr,                                                                                           \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "                                        \
                    "with "                                                                                           \
                    "%s (%d).\n",                                                                                     \
                    #call,                                                                                            \
                    __LINE__,                                                                                         \
                    __FILE__,                                                                                         \
                    hipGetErrorString( status ),                                                                     \
                    status );                                                                                         \
    }
#endif  // CUDA_RT_CALL
// *************** FOR ERROR CHECKING *******************

/*
* Factorial required for combinatorial number system
*/
constexpr size_t factorial( const int &n ) {
    return ( n <= 1 ) ? 1 : ( n * factorial( n - 1 ) );
}

/*
* The more we can do at compile time the better
*/
constexpr uint k_tpb { 512 };
constexpr uint k_ept { 1 };
constexpr uint k_numNodes { 10 };
constexpr uint k_numEdges { 9 };
constexpr uint k_numVertices { ( k_numNodes * ( k_numNodes - 1 ) ) >> 1 };
constexpr uint k_treeStart { k_numVertices - 1 };

/*
* Structure hold edge and angle information
*/
typedef struct fEdgeData_t {
    int a {};
    int b {};
} edgeData;

/*
* Structure to hold all combos and scores for testing
*/
typedef struct fgpuData_t {
    uint         offset {};
    uint         treesMaxDevice {};
    uint         treesPerDevice {};
    hipStream_t streams {};
} gpuData;

/*
* Structure to hold all combos and scores for testing
*/
typedef struct fCombos_t {
    uchar *d_treeCombos {};
    uchar *d_treeScores {};
} comboData;

/*
* Constant memory holds read-only data in cached global memory
*/
__constant__ double c_denominator[k_numEdges];
__constant__ edgeData c_edges[k_numVertices];

/*
* Device memory to hold total trees per device
*/
__device__ unsigned long long int d_totalTreesPerBlock {};

/*
* Calculate binomial coefficients
* Care must be taken to ensure arithmetic doesn't
* exceed what a data type can hold
*/
__host__ __device__ size_t nchoosek( const uint &numerator, const double &denominator, const int &loops ) {

    size_t n { static_cast<size_t>( numerator ) };
    for ( int f = 1; f < loops; f++ )
        n *= ( numerator - f );
    return ( static_cast<size_t>( static_cast<double>( n ) * denominator ) );  // Precalculate
}

/*
* This function is find a combination based on a given id
* It uses the combinatorial number system and produces
* an answer in lexicographic order
*/
__host__ __device__ void getTree( const uint &maxTrees, const uint &id, uint *combo ) {

    uint n {};
    uint key { maxTrees - id - 1 };
#pragma unroll k_numEdges
    for ( int e = 0; e < k_numEdges; e++ ) {
        uint numerator { k_treeStart };
        while ( true ) {
            // The denominator must start at the end of the array
            n = nchoosek( numerator, c_denominator[e], ( k_numEdges - e ) );
            if ( n <= key ) {
                combo[e] = k_treeStart - numerator;
                key -= n;
                break;
            }
            numerator--;
        }
    }
}

/*
* This function is used by Kruskal's Minimum Spanning Tree algorithm
*/
__device__ uint find( const uint &x, uint *parent ) {

    if ( parent[x] != x )
        parent[x] = find( parent[x], parent );
    return ( parent[x] );
}

/*
* This function is used to generate a sequence on the device
*/
__device__ void gen_seq( uint *parent ) {
#pragma unroll  k_numNodes
    for ( int i = 0; i < k_numNodes; i++ ) {
        parent[i] = i;
    }
}

/*
* CUDA kernel to determine minimum angle diversity score for a given block.
* Since we are using CUB BlockRadixSort and grid-stride looping
* we need to pad the last block in the last grid
*/
__launch_bounds__( k_tpb ) __global__ void tdoa( const uint offset,
                                                const uint treesPerDevice,
                                                const uint maxTrees,
                                                const uint padding,
                                                uchar *__restrict__ d_holdTrees,
                                                uchar *__restrict__ d_holdScore ) {

    const auto block { cooperative_groups::this_thread_block( ) };

    // Specialize BlockRadixSort for a 1D block of k_tpb threads of type int
    typedef hipcub::BlockReduce<uint, k_tpb> BlockReduce;

    // Allocate shared memory for BlockRadixSort
    __shared__ typename BlockReduce::TempStorage temp_storage;

    uint gid { blockIdx.x * blockDim.x + threadIdx.x };  // Change name
    uint stride { blockDim.x * gridDim.x };
    uint newTid[k_ept] {};

    for ( uint tid = gid; tid < padding; tid += stride ) {

        uint score[k_ept] {};

        // To increase Instruction Level Parallelism (ILP)
        // Each thread will calculate multiple combination scores
        for ( int s = 0; s < k_ept; s++ ) {
            newTid[s] = offset + tid * k_ept + s;

            // Ensure only valid combinations are checked
            if ( newTid[s] < treesPerDevice ) {

                // Find tree
                uint combo[k_numEdges] {};
                getTree( maxTrees, newTid[s], combo );

                // Determine is chain is valid spanning tree
                // Use Kruskal's algorithm with each edge weight set to 1
                uint parent[k_numNodes] {};
                gen_seq( parent );

                uint cost {};
#pragma unroll k_numEdges
                for ( int e = 0; e < k_numEdges; e++ ) {
                    uint findX = find( c_edges[combo[e]].a, parent );
                    uint findY = find( c_edges[combo[e]].b, parent );
                    if ( findX == findY )
                        continue;
                    cost++;
                    parent[findX] = findY;
                }

                // If the minimum spanning tree has a cost
                // less than the number of edges, than it
                // is not a valid spanning tree
                if ( cost == k_numEdges ) {

                    // Copy to global -> host to check for cycle
#pragma unroll k_numEdges
                    for ( int e = 0; e < k_numEdges; e++ )
                        d_holdTrees[( newTid[s] - offset ) * k_numEdges + e] = combo[e];

                    d_holdScore[newTid[s] - offset] = 1u;

                    score[s] = 1u;
                }
            }
        }

        uint totalTrees { BlockReduce( temp_storage ).Sum( score ) };

        // Once BlockReduce is finished, the sum
        // is now stored in the first address of the score array
        // in the first thread in the block. That value is
        // then stored to global memory to the address pertaining
        // to that blockId
        if ( !block.thread_rank( ) )
            atomicAdd( &d_totalTreesPerBlock, totalTrees );

        // We need to sync the block again because we are using grid-stride looping.
        block.sync( );  // Sync block to reuse tempStorage for BlockReduce
    }
}

int main( int arg, char **argv ) {

    // Determine the number of combinations that will be evaluated
    size_t maxTrees { nchoosek( k_numVertices, ( 1.0 / factorial( k_numEdges ) ), k_numEdges ) };
    if ( maxTrees >= UINT_MAX ) {  // maxTrees can't be larger than 4294967295
        std::printf( "combos = %d; chains = %lu\n", k_numVertices, maxTrees );
        throw std::runtime_error( "The number is chains to test is larger than uint can hold.\n" );
    }
    std::printf( "Number of Vertices = %d\n", k_numVertices );
    std::printf( "Max Trees Possible = %lu\n", maxTrees );

    // Precompute all possible denominators recipicals
    // Multiplication requires less operations than division
    double denominator[k_numEdges] {};
    for ( int i = k_numEdges; i > 0; i-- )
        denominator[k_numEdges - i] = 1.0 / factorial( i );

    // Generate all possible edges of graph in lexicographic order
    edgeData edges[k_numVertices] {};
    int      start { 1 };
    int      idx {};
    for ( int i = 0; i < k_numNodes; i++ ) {
        for ( int j = start; j < k_numNodes; j++ ) {
            edges[idx].a = i;
            edges[idx].b = j;
            idx++;
        }
        start++;
    }

    // Get device attributes
    int numDevices {};
    int numSMs {};
    CUDA_RT_CALL( hipGetDeviceCount( &numDevices ) );
    CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );
    std::printf( "Number of GPUs = %d\n", numDevices );

    // Padding blocks so we can use CUB BlockReduce in CUDA kernel
    uint padding { static_cast<uint>( std::ceil( static_cast<double>( maxTrees ) / k_ept / numDevices / k_tpb ) *
                                    k_tpb ) };

    // Will store final results for each GPU using pinned memory
    // Pinned memory is required for async copies
    thrust::host_vector<uint, thrust::cuda::experimental::pinned_allocator<uint>> h_totalTrees( numDevices, 0 );

    thrust::host_vector<uchar, thrust::cuda::experimental::pinned_allocator<uchar>> h_holdTrees( maxTrees * k_numEdges,
                                                                                                0 );

    thrust::host_vector<uchar, thrust::cuda::experimental::pinned_allocator<uchar>> h_holdScore( maxTrees, 0 );

    gpuData   gpuWork[numDevices] {};
    comboData comboStruct[numDevices] {};

    // Divide up work between all GPUs
    uint chunk { static_cast<uint>( maxTrees / numDevices ) };

    for ( int d = 0; d < numDevices; d++ ) {
        if ( d < ( numDevices - 1 ) ) {
            gpuWork[d].offset         = d * chunk;
            gpuWork[d].treesMaxDevice = ( d + 1 ) * chunk;
            gpuWork[d].treesPerDevice = chunk;
        } else {
            gpuWork[d].offset         = d * chunk;
            gpuWork[d].treesMaxDevice = maxTrees;
            gpuWork[d].treesPerDevice = maxTrees - d * chunk;
        }
    }

    // Launch one CPU thread per GPU
    omp_set_num_threads( numDevices );
#pragma omp parallel
    {
        int ompId { omp_get_thread_num( ) };

        // We must set the device in each thread
        // so the correct CUDA context is visible
        CUDA_RT_CALL( hipSetDevice( ompId ) );
        CUDA_RT_CALL( hipStreamCreate( &gpuWork[ompId].streams ) );

        // Allocate memory to store all combos for cyclic testing
        CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &comboStruct[ompId].d_treeCombos ),
                                sizeof( uchar ) * gpuWork[ompId].treesPerDevice * k_numEdges ) );
        CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &comboStruct[ompId].d_treeScores ),
                                sizeof( uchar ) * gpuWork[ompId].treesPerDevice ) );

        // Copy denominators to constant memory
        CUDA_RT_CALL( hipMemcpyToSymbolAsync(HIP_SYMBOL( c_denominator),
                                            denominator,
                                            sizeof( double ) * k_numEdges,
                                            0,
                                            hipMemcpyHostToDevice,
                                            gpuWork[ompId].streams ) );

        // Copy angles to constant memory
        CUDA_RT_CALL( hipMemcpyToSymbolAsync(HIP_SYMBOL(
            c_edges), edges, sizeof( edgeData ) * k_numVertices, 0, hipMemcpyHostToDevice, gpuWork[ompId].streams ) );
    }

    // Start timer
    hipEvent_t startEvent { nullptr };
    hipEvent_t stopEvent { nullptr };
    float       elapsed_gpu_ms {};

    CUDA_RT_CALL( hipEventCreate( &startEvent, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &stopEvent, hipEventBlockingSync ) );

    CUDA_RT_CALL( hipEventRecord( startEvent ) );

#pragma omp parallel
    {
        int ompId { omp_get_thread_num( ) };
        CUDA_RT_CALL( hipSetDevice( ompId ) );

        // The number of blocks launched is based on the number of
        // Streaming Multiprocessor available on the GPU
        int threadPerBlock { k_tpb };
        int blocksPerGrid { numSMs * 32 };

        void *args[] { &gpuWork[ompId].offset,           &gpuWork[ompId].treesMaxDevice,  &maxTrees, &padding,
                    &comboStruct[ompId].d_treeCombos, &comboStruct[ompId].d_treeScores };

        CUDA_RT_CALL( hipLaunchKernel(
            reinterpret_cast<void *>( &tdoa ), blocksPerGrid, threadPerBlock, args, 0, gpuWork[ompId].streams ) );

        // Scores and ids are copied back to the CPU in parallel
        CUDA_RT_CALL( hipMemcpyFromSymbolAsync( &h_totalTrees[ompId],
                                                HIP_SYMBOL(d_totalTreesPerBlock),
                                                sizeof( uint ),
                                                0,
                                                hipMemcpyDefault,
                                                gpuWork[ompId].streams ) );

        // Copy back combos to check for cycles
        CUDA_RT_CALL( hipMemcpyAsync( &h_holdTrees[gpuWork[ompId].offset * k_numEdges],
                                    comboStruct[ompId].d_treeCombos,
                                    sizeof( uchar ) * gpuWork[ompId].treesPerDevice * k_numEdges,
                                    hipMemcpyDeviceToHost,
                                    gpuWork[ompId].streams ) );

        CUDA_RT_CALL( hipMemcpyAsync( &h_holdScore[gpuWork[ompId].offset],
                                    comboStruct[ompId].d_treeScores,
                                    sizeof( uchar ) * gpuWork[ompId].treesPerDevice,
                                    hipMemcpyDeviceToHost,
                                    gpuWork[ompId].streams ) );

        // Sync each stream to ensure data copy is complete
        CUDA_RT_CALL( hipStreamSynchronize( gpuWork[ompId].streams ) );
    }

    // Stop timer
    CUDA_RT_CALL( hipEventRecord( stopEvent ) );
    CUDA_RT_CALL( hipEventSynchronize( stopEvent ) );

    CUDA_RT_CALL( hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent ) );
    std::printf( "Runtime = %0.2f ms\n\n", elapsed_gpu_ms );

    size_t numCyclics {};
    size_t tempIdx {};
    std::printf( "Verifying Results\n" );
#pragma omp parallel for num_threads( omp_get_max_threads( ) ) shared( numCyclics ) private( tempIdx )
    for ( size_t i = 0; i < maxTrees; i++ ) {
        if ( h_holdScore[i] ) {
            Graph g( k_numNodes );
            for ( int j = 0; j < k_numEdges; j++ ) {
                tempIdx = i * k_numEdges + j;
                g.addEdge( edges[h_holdTrees[tempIdx]].a, edges[h_holdTrees[tempIdx]].b );
            }
            if ( g.isCyclic( ) )
#pragma omp atomic
                numCyclics++;
        }
    }

    uint h_total { std::accumulate( h_totalTrees.begin( ), h_totalTrees.end( ), 0u ) };

    if ( ( k_numNodes - 1 ) == k_numEdges ) {
        std::printf( "%u trees found\n", h_total );
        uint cayley =
            static_cast<uint>( std::pow( static_cast<double>( k_numNodes ), static_cast<double>( k_numNodes - 2 ) ) );

        if ( h_total == cayley )
            std::printf( "Total trees equals Cayley's formula.\nFound correct number of trees!!\n" );
        else
            std::printf( "Error!!\n" );
    } else
        std::printf( "%u forests found\n", h_total );
    std::printf( "%lu cyclics found!\n", numCyclics );

    // Data clean up
#pragma omp parallel
    {
        int ompId { omp_get_thread_num( ) };
        CUDA_RT_CALL( hipSetDevice( ompId ) );
        CUDA_RT_CALL( hipFree( comboStruct[ompId].d_treeCombos ) );
        CUDA_RT_CALL( hipFree( comboStruct[ompId].d_treeScores ) );
        CUDA_RT_CALL( hipStreamDestroy( gpuWork[ompId].streams ) );
    }

    return ( EXIT_SUCCESS );
}
 