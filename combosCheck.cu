#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

#include <cmath>
#include <hip/hip_cooperative_groups.h>
#include <cub/block/block_reduce.cuh>
#include <hip/hip_runtime.h>
#include <numeric>
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

#include "graph.h"

// *************** FOR ERROR CHECKING *******************
#ifndef CUDA_RT_CALL
#define CUDA_RT_CALL( call )                                                                                           \
    {                                                                                                                  \
        auto status = static_cast<hipError_t>( call );                                                                \
        if ( status != hipSuccess )                                                                                   \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "                                        \
                     "with "                                                                                           \
                     "%s (%d).\n",                                                                                     \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     hipGetErrorString( status ),                                                                     \
                     status );                                                                                         \
    }
#endif  // CUDA_RT_CALL
// *************** FOR ERROR CHECKING *******************

/*
 * Factorial required for combinatorial number system
 */
constexpr double factorial( const int &n ) {
    return ( n <= 1 ) ? 1 : ( n * factorial( n - 1 ) );
}

/*
 * The more we can do at compile time the better
 */
constexpr unsigned int k_tpb { 512 };
constexpr int          k_ept { 8 };
constexpr int          k_numNodes { 9 };
constexpr int          k_numEdges { 8 };
constexpr int          k_numVertices { ( k_numNodes * ( k_numNodes - 1 ) ) / 2 };
constexpr int          k_treeStart { k_numVertices - 1 };

/*
 * Structure hold edge and angle information
 */
typedef struct fEdgeData_t {
    int a {};
    int b {};
} edgeData;

/*
 * Structure to hold all combos and scores for testing
 */
typedef struct fgpuData_t {
    unsigned int  offset {};
    unsigned int  treesMaxDevice {};
    unsigned int  treesPerDevice {};
    unsigned int *d_totalTreesPerBlock {};
    hipStream_t  streams {};
} gpuData;

/*
 * Structure to hold all combos and scores for testing
 */
typedef struct fCombos_t {
    unsigned int *d_treeCombos {};
    unsigned int *d_treeScores {};
} comboData;

/*
 * Constant memory holds read-only data in cached global memory
 */
__constant__ double c_denominator[k_numEdges];
__constant__ edgeData c_edges[k_numVertices];

/*
 * Calculate binomial coefficients
 * Care must be taken to ensure arithmetic doesn't
 * exceed what a data type can hold
 */
__host__ __device__ unsigned int nchoosek( const int &numerator, const double &denominator, const int &loops ) {

    unsigned long long n { static_cast<unsigned long long>( numerator ) };
    for ( int f = 1; f < loops; f++ )
        n *= static_cast<unsigned long long>( numerator - f );
    return ( static_cast<unsigned int>( static_cast<double>( n ) * denominator ) );  // Precalculate
}

/*
 * This function is find a combination based on a given id
 * It uses the combinatorial number system and produces
 * an answer in lexicographic order
 */
__host__ __device__ void
         getTree( const unsigned int &maxTrees, const unsigned int &id, const double *denominator, int *combo ) {

    unsigned int n {};
    unsigned int key { maxTrees - id - 1 };
#pragma unroll k_numEdges
    for ( int e = 0; e < k_numEdges; e++ ) {
        int numerator { k_treeStart };
        while ( true ) {
            // The denominator must start at the end of the array
            n = nchoosek( numerator, denominator[e], ( k_numEdges - e ) );
            if ( n <= key ) {
                combo[e] = k_treeStart - numerator;
                key -= n;
                break;
            }
            numerator--;
        }
    }
}

/*
 * This function is used by Kruskal's Minimum Spanning Tree algorithm
 */
__device__ int find( const int &x, int *parent ) {

    if ( parent[x] != x )
        parent[x] = find( parent[x], parent );
    return ( parent[x] );
}

/*
 * CUDA kernel to determine minimum angle diversity score for a given block.
 * Since we are using CUB BlockRadixSort and grid-stride looping
 * we need to pad the last block in the last grid
 */
__launch_bounds__( k_tpb ) __global__ void tdoa( const unsigned int offset,
                                                 const unsigned int treesPerDevice,
                                                 const unsigned int maxTrees,
                                                 const unsigned int padding,
                                                 unsigned int *__restrict__ d_totalTreesPerBlock,
                                                 unsigned int *__restrict__ d_holdTrees,
                                                 unsigned int *__restrict__ d_holdScore ) {

    const auto block { cooperative_groups::this_thread_block( ) };

    // Specialize BlockRadixSort for a 1D block of k_tpb threads of type int
    typedef hipcub::BlockReduce<unsigned int, k_tpb> BlockReduce;

    // Allocate shared memory for BlockRadixSort
    __shared__ typename BlockReduce::TempStorage temp_storage;

    unsigned int gid { blockIdx.x * blockDim.x + threadIdx.x };  // Change name
    unsigned int stride { blockDim.x * gridDim.x };
    unsigned int newTid[k_ept] {};
    unsigned int score[k_ept] {};

    for ( unsigned int tid = gid; tid < padding; tid += stride ) {

        // To increase Instruction Level Parallelism (ILP)
        // Each thread will calculate multiple combination scores
        for ( int s = 0; s < k_ept; s++ ) {
            newTid[s] = offset + tid * k_ept + s;

            // Ensure only valid combinations are checked
            if ( newTid[s] < treesPerDevice ) {

                // Find tree
                int combo[k_numEdges] {};
                getTree( maxTrees, newTid[s], c_denominator, combo );

                // Determine is chain is valid spanning tree
                // Use Kruskal's algorithm with each edge weight set to 1
                // thrust::seq allows us to run Thrust functions
                // in individual threads
                int parent[k_numNodes] {};
                thrust::sequence( thrust::seq, parent, parent + k_numNodes, 0 );

                int cost {};
#pragma unroll k_numEdges
                for ( int e = 0; e < k_numEdges; e++ ) {
                    int findX = find( c_edges[combo[e]].a, parent );
                    int findY = find( c_edges[combo[e]].b, parent );
                    if ( findX == findY )
                        continue;
                    cost++;
                    parent[findX] = findY;
                }

                // If the minimum spanning tree has a cost
                // less than the number of edges, than it
                // is not a valid spanning tree
                if ( cost == k_numEdges ) {

                    // Copy to global -> host to check for cycle
#pragma unroll k_numEdges
                    for ( int e = 0; e < k_numEdges; e++ )
                        d_holdTrees[( newTid[s] - offset ) * k_numEdges + e] = combo[e];

                    d_holdScore[( newTid[s] - offset )] = 1u;

                    score[s] = 1u;

                } else {
                    score[s] = 0u;
                }
            } else
                score[s] = 0u;  // For thread ids larger than the number of required combinations
        }

        unsigned int totalTrees { BlockReduce( temp_storage ).Sum( score ) };

        // Once BlockReduce is finished, the sum
        // is now stored in the first address of the score array
        // in the first thread in the block. That value is
        // then stored to global memory to the address pertaining
        // to that blockId
        if ( threadIdx.x == 0 )
            atomicAdd( &d_totalTreesPerBlock[0], totalTrees );

        // We need to sync the block again because we are using grid-stride looping.
        block.sync( );  // Sync block to reuse tempStorage for BlockReduce
    }
}

int main( int arg, char **argv ) {

    // Determine the number of combinations that will be evaluated
    unsigned int maxTrees { nchoosek( k_numVertices, ( 1 / factorial( k_numEdges ) ), k_numEdges ) };
    if ( maxTrees >= UINT_MAX ) {  // maxTrees can't be larger than 4294967295
        std::printf( "combos = %d; chains = %u\n", k_numVertices, maxTrees );
        throw std::runtime_error( "The number is chains to test is larger than unsigned int can hold.\n" );
    }
    std::printf( "Number of Vertices = %d\n", k_numVertices );
    std::printf( "Max Trees Possible = %u\n", maxTrees );

    // Precompute all possible denominators recipicals
    // Multiplication requires less operations than division
    double denominator[k_numEdges] {};
    for ( int i = k_numEdges; i > 0; i-- )
        denominator[k_numEdges - i] = 1 / factorial( i );

    // Generate all possible edges of graph in lexicographic order
    edgeData edges[k_numVertices] {};
    int      start { 1 };
    int      idx {};
    for ( int i = 0; i < k_numNodes; i++ ) {
        for ( int j = start; j < k_numNodes; j++ ) {
            edges[idx].a = i;
            edges[idx].b = j;
            idx++;
        }
        start++;
    }

    // Get device attributes
    int numDevices {};
    int numSMs {};
    CUDA_RT_CALL( hipGetDeviceCount( &numDevices ) );
    CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );
    std::printf( "Number of GPUs = %d\n", numDevices );

    // Padding blocks so we can use CUB BlockReduce in CUDA kernel
    unsigned int padding { static_cast<unsigned int>(
        std::ceil( static_cast<double>( maxTrees ) / static_cast<double>( k_ept ) / static_cast<double>( numDevices ) /
                   static_cast<double>( k_tpb ) ) *
        k_tpb ) };

    // Will store final results for each GPU using pinned memory
    // Pinned memory is required for async copies
    thrust::host_vector<unsigned int, thrust::cuda::experimental::pinned_allocator<unsigned int>> h_totalTrees(
        numDevices, 0 );

    thrust::host_vector<unsigned int, thrust::cuda::experimental::pinned_allocator<unsigned int>> h_holdTrees(
        maxTrees * k_numEdges, 0 );

    thrust::host_vector<unsigned int, thrust::cuda::experimental::pinned_allocator<unsigned int>> h_holdScore( maxTrees,
                                                                                                               0 );

    gpuData   gpuWork[numDevices] {};
    comboData comboStruct[numDevices] {};

    // Divide up work between all GPUs
    unsigned int chunk { static_cast<unsigned int>( maxTrees / numDevices ) };

    for ( int d = 0; d < numDevices; d++ ) {
        if ( d < ( numDevices - 1 ) ) {
            gpuWork[d].offset         = d * chunk;
            gpuWork[d].treesMaxDevice = ( d + 1 ) * chunk;
            gpuWork[d].treesPerDevice = chunk;
        } else {
            gpuWork[d].offset         = d * chunk;
            gpuWork[d].treesMaxDevice = maxTrees;
            gpuWork[d].treesPerDevice = maxTrees - d * chunk;
        }
    }

    // Launch one CPU thread per GPU
    omp_set_num_threads( numDevices );
#pragma omp parallel
    {
        int ompId { omp_get_thread_num( ) };

        // We must set the device in each thread
        // so the correct CUDA context is visible
        CUDA_RT_CALL( hipSetDevice( ompId ) );
        CUDA_RT_CALL( hipStreamCreate( &gpuWork[ompId].streams ) );

        // Allocate memory to hold total number of valid trees per block and device
        CUDA_RT_CALL(
            hipMalloc( reinterpret_cast<void **>( &gpuWork[ompId].d_totalTreesPerBlock ), sizeof( unsigned int ) ) );

        // Allocate memory to store all combos for cyclic testing
        CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &comboStruct[ompId].d_treeCombos ),
                                  gpuWork[ompId].treesPerDevice * k_numEdges * sizeof( unsigned int ) ) );
        CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &comboStruct[ompId].d_treeScores ),
                                  gpuWork[ompId].treesPerDevice * sizeof( unsigned int ) ) );

        // Copy denominators to constant memory
        CUDA_RT_CALL( hipMemcpyToSymbolAsync(HIP_SYMBOL( c_denominator),
                                               denominator,
                                               k_numEdges * sizeof( double ),
                                               0,
                                               hipMemcpyHostToDevice,
                                               gpuWork[ompId].streams ) );

        // Copy angles to constant memory
        CUDA_RT_CALL( hipMemcpyToSymbolAsync(HIP_SYMBOL(
            c_edges), edges, k_numVertices * sizeof( edgeData ), 0, hipMemcpyHostToDevice, gpuWork[ompId].streams ) );
    }

    // Start timer
    hipEvent_t startEvent { nullptr };
    hipEvent_t stopEvent { nullptr };
    float       elapsed_gpu_ms {};

    hipEventCreate( &startEvent, hipEventBlockingSync );
    hipEventCreate( &stopEvent, hipEventBlockingSync );

    hipEventRecord( startEvent );

#pragma omp parallel
    {
        int ompId { omp_get_thread_num( ) };
        CUDA_RT_CALL( hipSetDevice( ompId ) );

        // The number of blocks launched is based on the number of
        // Streaming Multiprocessor available on the GPU
        dim3 threadPerBlock { k_tpb };
        dim3 blocksPerGrid { static_cast<uint>( 20 * numSMs ) };

        void *args[] { &gpuWork[ompId].offset,
                       &gpuWork[ompId].treesMaxDevice,
                       &maxTrees,
                       &padding,
                       &gpuWork[ompId].d_totalTreesPerBlock,
                       &comboStruct[ompId].d_treeCombos,
                       &comboStruct[ompId].d_treeScores };

        CUDA_RT_CALL( hipLaunchKernel(
            reinterpret_cast<void *>( &tdoa ), blocksPerGrid, threadPerBlock, args, 0, gpuWork[ompId].streams ) );

        // Scores and ids are copied back to the CPU in parallel
        CUDA_RT_CALL( hipMemcpyAsync( &h_totalTrees[ompId],
                                       gpuWork[ompId].d_totalTreesPerBlock,
                                       sizeof( unsigned int ),
                                       hipMemcpyDeviceToHost,
                                       gpuWork[ompId].streams ) );

        // Copy back combos to check for cycles
        CUDA_RT_CALL( hipMemcpyAsync( &h_holdTrees[gpuWork[ompId].offset * k_numEdges],
                                       comboStruct[ompId].d_treeCombos,
                                       gpuWork[ompId].treesPerDevice * k_numEdges * sizeof( unsigned int ),
                                       hipMemcpyDeviceToHost ) );

        CUDA_RT_CALL( hipMemcpyAsync( &h_holdScore[gpuWork[ompId].offset],
                                       comboStruct[ompId].d_treeScores,
                                       gpuWork[ompId].treesPerDevice * sizeof( unsigned int ),
                                       hipMemcpyDeviceToHost ) );

        // Sync each stream to ensure data copy is complete
        CUDA_RT_CALL( hipStreamSynchronize( gpuWork[ompId].streams ) );
    }

    // Stop timer
    hipEventRecord( stopEvent );
    hipEventSynchronize( stopEvent );

    hipEventElapsedTime( &elapsed_gpu_ms, startEvent, stopEvent );
    std::printf( "Runtime = %0.2f ms\n", elapsed_gpu_ms );

    unsigned int numCyclics {};
    int          tempIdx {};
#pragma omp parallel for num_threads( omp_get_max_threads( ) ) shared( numCyclics ) private( tempIdx )
    for ( int i = 0; i < maxTrees; i++ ) {
        if ( h_holdScore[i] ) {
            Graph g( k_numNodes );
            for ( int j = 0; j < k_numEdges; j++ ) {
                tempIdx = i * k_numEdges + j;
                g.addEdge( edges[h_holdTrees[tempIdx]].a, edges[h_holdTrees[tempIdx]].b );
            }
            if ( g.isCyclic( ) )
#pragma omp atomic
                numCyclics++;
        }
    }

    unsigned int h_total { std::accumulate( h_totalTrees.begin( ), h_totalTrees.end( ), 0u ) };

    if ( ( k_numNodes - 1 ) == k_numEdges ) {
        std::printf( "%u trees found\n", h_total );
        unsigned int cayley = static_cast<unsigned int>(
            pow( static_cast<double>( k_numNodes ), static_cast<double>( k_numNodes - 2 ) ) );

        if ( h_total == cayley )
            std::printf( "Total trees equals Cayley's formula.\nFound correct number of trees!!\n" );
        else
            std::printf( "Error!!\n" );
    } else
        std::printf( "%u forests found\n", h_total );
    std::printf( "%u cyclics found!\n", numCyclics );

    // Data clean up
#pragma omp parallel
    {
        int ompId { omp_get_thread_num( ) };
        CUDA_RT_CALL( hipSetDevice( ompId ) );
        CUDA_RT_CALL( hipFree( gpuWork[ompId].d_totalTreesPerBlock ) );
        CUDA_RT_CALL( hipFree( comboStruct[ompId].d_treeCombos ) );
        CUDA_RT_CALL( hipFree( comboStruct[ompId].d_treeScores ) );
        CUDA_RT_CALL( hipStreamDestroy( gpuWork[ompId].streams ) );
    }

    return ( EXIT_SUCCESS );
}
